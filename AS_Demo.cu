
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#define N 50000
#define BLOCK_SIZE 512

//Given two indices in an array (representing atoms),
//calculate their product (potential energy),
//and store in energies array.
//Parallel
__global__ void calcEnergyParallel(int *atoms, int numAtoms, int *energies, int numEnergies)
{
	int atom1 = blockIdx.x, atom2 = blockIdx.y * blockDim.x + threadIdx.x,
		energyIdx;
	
	if (atom2 < numAtoms && atom2 > atom1)
	{
		energyIdx = gridDim.x * atom1 + atom2 - (blockIdx.x + 1) * (blockIdx.x + 2) / 2;
		energies[energyIdx] = atoms[atom1] * atoms[atom2];
	}
}

//Given two indices in an array (representing atoms),
//calculate their product (potential energy),
//and store in energies array.
//Serial
void calcEnergySerial(int *atoms, int numAtoms, int *energies, int numEnergies)
{
	int i, j, k;
	
	for (i = 0; i < numAtoms; i++)
	{
		for (j = 0; j < numAtoms; j++)
		{
			if (j > i)
			{
				k = N * i + j - (i + 1) * (i + 2) / 2;
				energies[k] = atoms[i] * atoms[j];
			}
		}
	}
}

int main()
{
	clock_t S_TIME, P_TIME;
	int *atomsHost, *atomsDevice, *energiesHost, *energiesDevice, gridYDim = 1, blockXDim = N;
	unsigned long int totalEnergy, atomsSize, energiesSize;
	
	atomsSize = N * sizeof(int);
	energiesSize = sizeof(int) * N * (N - 1) / 2;
	
	atomsHost = (int*) malloc(atomsSize);
	energiesHost = (int*) malloc(energiesSize);
	
	int i;
	for (i = 0; i < N; i++)
	{
		atomsHost[i] = i;
	}
	
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		energiesHost[i] = 0;
	}
	
	//Serial Run
	S_TIME = clock();
	calcEnergySerial(atomsHost, N,energiesHost, energiesSize / sizeof(int));

	totalEnergy = 0;
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		//totalEnergy += energiesHost[i];
	}
	
	printf("Serial: Total Energy for %u atoms is %u Pseudo-Joules.\n", N, totalEnergy);
	S_TIME = clock() - S_TIME;
	
	//Reset energiesHost
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		energiesHost[i] = 0;
	}

	//Parallel Run
	P_TIME = clock();
	if (N > BLOCK_SIZE)
	{
		gridYDim = N / BLOCK_SIZE + 1;
		blockXDim = BLOCK_SIZE;
	}
	dim3 gridDim(N, gridYDim, 1);
	dim3 blockDim(blockXDim, 1, 1);

	hipMalloc(&atomsDevice, atomsSize);
	hipMalloc(&energiesDevice, energiesSize);
	
	hipMemcpy(atomsDevice, atomsHost, atomsSize, hipMemcpyHostToDevice);
	hipMemcpy(energiesDevice, energiesHost, energiesSize, hipMemcpyHostToDevice);
	
	//N blocks of N threads (every atom pair)
	calcEnergyParallel<<<gridDim, blockDim>>>(atomsDevice, N, energiesDevice, energiesSize / sizeof(int));

	hipMemcpy(energiesHost, energiesDevice, energiesSize, hipMemcpyDeviceToHost);

	totalEnergy = 0;
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		//printf("%u: %u\n", i, energiesHost[i]);
		//totalEnergy += energiesHost[i];
	}
	
	printf("Parallel: Total Energy for %u atoms is %u Pseudo-Joules.\n", N, totalEnergy);
	P_TIME = clock() - P_TIME;

	printf("The parallel code runs %fx as fast as the serial version.\n", (float) S_TIME / (float) P_TIME);
	
	return 0;
}